#include "hip/hip_runtime.h"
#include "my_headers.hpp"
#include "my_gsl_utilities.hpp"
#include "my_matrices.hpp"
#include "var_utilities.hpp"
#include "var_pointmass.hpp"
#include "var_gaussian.hpp"
#include "var_gamma.hpp"
#include "var_pos_gaussian.hpp"
#include "emb_params.hpp"
#include "emb_io.hpp"
#include "emb_inference.hpp"

int main (int argc, char *argv[]) {
	my_param param;
	my_hyper hyper;
	my_data data;

	/******** Read options from input (argc, argv) ********/
	std::cout << "Initializing program..." << endl;
	my_input::read_input_from_command_line(argc,argv,data,param,hyper);

	/******** Read data from file ********/
	std::cout << "Reading data..." << endl;
	// Read train.tsv, validation.tsv and train.tsv
	my_input::read_data_file(data,param);
	// Create auxiliary data structures
	std::cout << "Computing auxiliary data structures..." << endl;
	data.create_transactions_train(param);
	data.create_sessions_per_user(param);
	data.create_other_data_structs(param);
	std::cout << "Computing auxiliary data structures for test set..." << endl;
	data.create_transactions_test(param);
	// Create output folder
	my_output::create_output_folder(data,param);
	// Print vocabulary
	my_output::write_vocab(data,param);
	// Read prices
	if(param.flag_price>0) {
		std::cout << "Reading prices..." << endl;
		my_input::read_price_file(data,param);
		my_input::normalize_prices(data,param);
		my_output::write_normalized_prices(data,param);
		my_input::take_log_prices(data,param);
	}
	// Read itemgroups
	if(param.flag_day>0 || param.flag_price>0 || param.Kgroup>0 || param.flag_nsFreq>1 || param.flag_likelihood==3) {
		// read itemGroup.tsv
		std::cout << "Reading item groups..." << endl;
		my_input::read_itemgroups(data,param);
	}
	// Read sess_days
	if(param.flag_day>0) {
		// read sess_days.tsv
		std::cout << "Reading session/days mapping..." << endl;
		my_input::read_sess_days(data,param);
	}
	// Create auxiliary data structures (continued)
	data.compute_unigram_distributions(param);

	/******** Write log file ********/
	my_output::create_log_file(data,param,hyper);

	/******** Set the seeds ********/
	srand48(param.seed);
	gsl_rng *semilla = gsl_rng_alloc(gsl_rng_taus);
	gsl_rng_set(semilla,param.seed);

	/******** Allocate memory for variational parameters ********/
	std::cout << "Initializing latent parameters..." << endl;
	my_pvar pvar(data,param);
	// Initialize variational parameters
	pvar.initialize_all(semilla,data,param,hyper);

	/******** Inference algorithm ********/
	std::cout << "Running inference algorithm..." << endl;
	time_t t_ini_abs = time(NULL);
	time_t t_ini;
	time_t t_end;
	bool stop = false;
	param.it = 0;
	int why = -1;
	double val_llh;
	int duration;
	while(!stop) {
		std::cout << " +Iteration " << param.it << "..." << endl;
		t_ini = time(NULL);	// Measure time elapsed

		// Run an inference step
		my_infer::inference_step(data,hyper,param,pvar,semilla);

		// Write time elapsed
		t_end = time(NULL);	// Measure time elapsed
		my_output::write_telapsed(param.outdir,param.it,t_ini,t_end);
		duration = difftime(t_end,t_ini_abs);

		// Compute and write norm
		double auxNormTheta = 0.0;
		double auxNormRho = 0.0;
		if(param.flag_userVec>0) {
			auxNormTheta = my_infer::compute_avg_norm(pvar.theta);
		}
		if(!param.flag_symmetricRho) {
			auxNormRho = my_infer::compute_avg_norm(pvar.rho);
		}
		my_output::write_norm(param.outdir,param.it,auxNormRho,my_infer::compute_avg_norm(pvar.alpha),auxNormTheta);

		// Compute llh + Check stop criteria
		if((param.it%param.rfreq)==0 && param.it>0) {
			my_infer::prepare_test_valid(semilla,data,hyper,param,pvar);
			val_llh = my_infer::compute_val_likelihood(false,duration,data,param,hyper,pvar);
			my_infer::compute_test_performance(false,duration,data,hyper,param,pvar);
			if(!param.noVal && (param.it>3*param.rfreq)) {
				std::cout << " (validation llh relative change (abs value): " << fabs((val_llh-param.prev_val_llh)/param.prev_val_llh) << ")" << endl;
                /*
				// If log likelihood increased, is not zero, and it increased less than 0.000001 of the previous value, set why to zero
				if(val_llh>=param.prev_val_llh && param.prev_val_llh!=0 && fabs((val_llh-param.prev_val_llh)/param.prev_val_llh)<param.valTolerance) {
					stop = true;
					why = 0;
				} else if(val_llh<param.prev_val_llh) {
					// Count the number of times in a row that the likelihood decreased
					(param.n_val_decr)++;
				} else if(val_llh>param.prev_val_llh) {
					param.n_val_decr = 0;
				}
				if(param.n_val_decr>param.valConsecutive) {
					stop = true;
					why = 1;
				}
                */
			}
			param.prev_val_llh = val_llh;
		}
		if(param.it+1>=param.Niter) {
			stop = true;
			why = 2;
		}
		if(!stop) {
			(param.it)++;
			// Save results to file if desired
			if((param.it%param.saveCycle)==0) {
				my_output::write_all_matrices("_it"+std::to_string(param.it),data,hyper,param,pvar);
			}
		}
	}

	/******** Print output ********/
    // Print output matrices
    my_output::write_all_matrices("",data,hyper,param,pvar);
	// Compute final validation llh & Print max file
	my_infer::prepare_test_valid(semilla,data,hyper,param,pvar);
	val_llh = my_infer::compute_val_likelihood(true,duration,data,param,hyper,pvar);
	my_output::write_max_file(param,duration,val_llh,why);
	my_infer::compute_test_performance(true,duration,data,hyper,param,pvar);
	my_infer::compute_test_performance_baskets(true,duration,data,hyper,param,pvar);

	/******** Free memory ********/
	gsl_rng_free(semilla);

	/******** Return ********/
	return 0;
};

